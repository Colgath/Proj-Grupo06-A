#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <stdio.h>

#include <opencv2/opencv.hpp>
//#include <opencv2/core/cuda.hpp>


// #include <opencv/cv.h> //Minha versão é outra.
// #include <opencv/ml.h>
// #include <opencv/cxcore.h>
// #include <opencv/highgui.h>


#include <sys/time.h> //Não existe no Windows.


#include <hip/hip_runtime.h> //Original

//#include <CUDA/hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void smoothGray (unsigned char *imagem, unsigned char *saida, unsigned int cols, unsigned int linhas)
{
	unsigned int indice = (blockIdx.y * blockDim.x * 65536) + (blockIdx.x * 1024) + threadIdx.x; // calcula o indice do vetor com base nas dimensões de bloco e indice da thread
	if(indice >= cols*linhas)
		return;
	//indices para o campo da imagem que participará do smooth 
	int i_begin = (indice/(int)cols) - 2, i_end = (indice/(int)cols)+2;
	int j_begin = (indice%(int)cols) - 2, j_end = (indice%(int)cols)+2;
	if(i_begin<0) i_begin = 0;
	if(j_begin<0) j_begin = 0;
	if(i_end>=cols) i_end = cols-1;
	if(j_end>=cols) j_end = cols-1;
	
	//calcula o smooth no ponto de indice da thread
	int media = 0;
	int qtd = 0;
	for (int i = i_begin; i<=  i_end; ++i)
	{
		for(int j = j_begin; j<= j_end; ++j)
		{
			media += imagem[(i*cols)+j];
			qtd++;
		}
	}
	saida[indice] = (unsigned char)(media/qtd);
}

void cudaCinza(char *nome_imagem, char *nome_saida)
{
	cv::Mat imagem = cv::imread(nome_imagem, CV_LOAD_IMAGE_GRAYSCALE); //abre a imagem de origem
	cv::Mat saida(imagem.rows, imagem.cols, imagem.type()); //cria a imagem de destino

	struct timeval start,end;
    double tempo=0.0;
    gettimeofday(&start,NULL);
	
	unsigned char * imagem_entrada;
	unsigned char * imagem_saida;
	hipMalloc((void **)&(imagem_entrada), sizeof(unsigned char) * imagem.rows*imagem.cols);//aloca a imagem de origem na GPU
	hipMemcpy((void *)(imagem_entrada), (void *)(imagem.data), sizeof(unsigned char) * imagem.rows * imagem.cols, hipMemcpyHostToDevice); //manda a imagem de origem pra GPU
	
	hipMalloc((void **)&(imagem_saida), sizeof(unsigned char) * imagem.rows*imagem.cols);//aloca a imagem de destino na GPU
		
	/* Inicio do processo paralelo */
	
//	hipInit(0); 
	dim3 Bloco_dim(1024); //define bloco de 1 linha tamanho 1024
	unsigned int num_Blocos = (unsigned int)ceil(((double)(imagem.rows*imagem.cols))/1024); //verifica quantos blocos serão necessário para a imagem
	smoothGray <<< num_Blocos, Bloco_dim >>> (imagem_entrada, imagem_saida, imagem.cols, imagem.rows);
	hipDeviceSynchronize(); 
	/* Fim do processo paralelo */
	
	hipMemcpy((void *)(saida.data), imagem_saida,(size_t)(sizeof(unsigned char) * imagem.rows * imagem.cols), hipMemcpyDeviceToHost); //copia a imagem resultante para o Host
	
	gettimeofday(&end,NULL);
    tempo =( ((double) ( ((end.tv_sec * 1000000 + end.tv_usec)
                                - (start.tv_sec * 1000000 + start.tv_usec))))/1000000);
    char arquivo[100];

    sprintf(arquivo, "%s.out", nome_saida);
    FILE *fp = NULL;
    if((fp = fopen(arquivo, "a")) == NULL)
		fp = fopen(arquivo,"w");
   
    fprintf(fp, "%lf\n", tempo);
    //apresentar os resultado
    fclose(fp);
	
	cv::imwrite(nome_saida, saida); //Escreve imagem no arquivo
	
	hipFree(imagem_entrada);
	hipFree(imagem_saida);
}

__global__ void smoothColor (unsigned char *imagem, unsigned char *saida, unsigned int cols, unsigned int linhas)
{
	unsigned int indice = (blockIdx.y * blockDim.x * 65536) + (blockIdx.x * 1024) + threadIdx.x; // calcula o indice do vetor com base nas dimensões de bloco e indice da thread
	if(indice >= cols*linhas)
		return;
	//indices para o campo da imagem que participará do smooth 
	int i_begin = (indice/(int)cols)-2, i_end = (indice/(int)cols)+2;
	int j_begin = (indice%(int)cols)-2, j_end = (indice%(int)cols)+2;
	if(i_begin<0) i_begin = 0;
	if(j_begin<0) j_begin = 0;
	if(i_end>=cols) i_end = cols-1;
	if(j_end>=cols) j_end = cols-1;
	
	//calcula o smooth no ponto de indice da thread
	int media[3] = {0,0,0};
	int qtd = 0;
	for (int i = i_begin; i<=  i_end; ++i)
	{
		for(int j = j_begin; j<= j_end; ++j)
		{
			media[0] += imagem[((i*cols)+j)*3];
			media[1] += imagem[((i*cols)+j)*3+1];
			media[2] += imagem[((i*cols)+j)*3+2];
			qtd++;
		}
	}

	saida[indice*3] = (unsigned char)(media[0]/qtd);
	saida[indice*3+1] = (unsigned char)(media[1]/qtd);
	saida[indice*3+2] = (unsigned char)(media[2]/qtd);
}

void cudaColorido(char *nome_imagem, char *nome_saida)
{
	cv::Mat imagem = cv::imread(nome_imagem); //abre a imagem de origem
	cv::Mat saida(imagem.rows, imagem.cols, CV_8UC3); //cria a imagem de destino
	
	struct timeval start,end;
    double tempo=0.0;
    gettimeofday(&start,NULL);
	
	unsigned char * imagem_entrada;
	unsigned char * imagem_saida;
	hipMalloc((void **)&(imagem_entrada), sizeof(unsigned char) * imagem.rows*imagem.cols * 3);//aloca a imagem de origem na GPU
	hipMemcpy((void *)(imagem_entrada), (void *)(imagem.data), sizeof(unsigned char) * imagem.rows * imagem.cols * 3, hipMemcpyHostToDevice); //manda a imagem de origem pra GPU

	hipMalloc((void **)&(imagem_saida), sizeof(unsigned char) * imagem.rows*imagem.cols * 3);//aloca a imagem de destino na GPU
	
	/* Inicio do processo paralelo */
	
	//hipInit(0); 
	dim3 Bloco_dim(1024); //define bloco de 1 linha tamanho 1024

	unsigned int num_Blocos = (unsigned int)ceil(((double)(imagem.rows*imagem.cols)) / 1024); //verifica quantos blocos serão necessário para a imagem

	smoothColor << < num_Blocos, Bloco_dim >> > (imagem_entrada, imagem_saida, imagem.cols, imagem.rows); 
	hipDeviceSynchronize(); 

	/* Fim do processo paralelo */
	

	hipMemcpy((void *)(saida.data), (void *)imagem_saida, (size_t)(sizeof(unsigned char) * imagem.rows * imagem.cols * 3), hipMemcpyDeviceToHost); //copia a imagem resultante para o Host
	
	gettimeofday(&end,NULL);
    tempo =( ((double) ( ((end.tv_sec * 1000000 + end.tv_usec)
                                - (start.tv_sec * 1000000 + start.tv_usec))))/1000000);
	
	char arquivo[100];
   //char *nome = strcat (nome_saida, ".out");
    sprintf(arquivo, "%s.out", nome_saida);
    FILE *fp = NULL;
    if((fp = fopen(arquivo, "a")) == NULL)
		fp = fopen(arquivo,"w");
   
    fprintf(fp, "%lf\n", tempo);
    //apresentar os resultado
    fclose(fp);
	
	
	cv::imwrite(nome_saida, saida); //Escreve imagem no arquivo
	
	hipFree(imagem_entrada);
	hipFree(imagem_saida);
}

int main()
{

	int opcao_cor=0;
	
	char nome_imagem[100];
	char nome_saida[100];
    std::cout << "Projeto Final-----Concorrentes-----\n" ;
    std::cout << "Eduardo Brunaldi dos Santos & Igor de Souza Baliza\n";
    std::cout << "\n\n\n";
	do{
		std::cout << "Escolha o tipo de Imagem:\n";
		std::cout << "0 - Sair\n";
		std::cout << "1 - RGB\n";
		std::cout << "2 - Grayscale\n";
		std::cout << "Digite uma das opcoes: ";
		std::cin >> opcao_cor;
		
		if(opcao_cor != 0)
		{
			int indice=0, i;
			char buffer_saida[100];
			std::cout << "Digite o nome do arquivo de entrada: ";
			std::cin >> nome_imagem;
			
			for(i=0; i < strlen(nome_imagem); ++i)
			{
				if(nome_imagem[i] == '/' ) indice = i+1;
			}
			
				int j = 0;
				for(; j <= i-indice; ++j)
				{
					buffer_saida[j] = nome_imagem[indice+j];
				}
			
			sprintf (nome_saida, "saida/%s", buffer_saida);
			if(opcao_cor == 1){
				cudaColorido(nome_imagem, nome_saida);
			}else{
				cudaCinza(nome_imagem, nome_saida);
			}
			break;

		}
		
	}while(opcao_cor != 0);


    return 0;
}
